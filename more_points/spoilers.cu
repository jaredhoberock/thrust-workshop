#include "hip/hip_runtime.h"
#include <thrust/device_vector.h>
#include <thrust/tabulate.h>
#include <thrust/random.h>
#include <thrust/transform.h>
#include <thrust/transform_scan.h>
#include <thrust/sort.h>
#include <thrust/reduce.h>
#include <thrust/binary_search.h>
#include <thrust/iterator/constant_iterator.h>
#include <iostream>
#include <iomanip>
#include <bitset>
#include <algorithm>
#include "util.h"

// Markers
enum { NODE = 1, LEAF = 2, EMPTY = 4 };

// Utility functions to encode leaves and children in single int
// are defined in util.h:
//   bool is_empty(int id);
//   bool is_node(int id);
//   bool is_leaf(int id);
//   int get_empty_id();
//   int get_leaf_id(int offset);
//   int get_leaf_offset(int id);

// Operator which merges two bounding boxes.
struct merge_bboxes
{
  inline __host__ __device__
  bbox operator()(const bbox &b0, const bbox &b1) const
  {
    bbox bounds;
    bounds.xmin = min(b0.xmin, b1.xmin);
    bounds.xmax = max(b0.xmax, b1.xmax);
    bounds.ymin = min(b0.ymin, b1.ymin);
    bounds.ymax = max(b0.ymax, b1.ymax);
    return bounds;
  }
};

bbox compute_bounding_box(const thrust::device_vector<float2> &points)
{
  return thrust::reduce(points.begin(), points.end(), bbox(), merge_bboxes());
}


// Classify a point with respect to the bounding box.
struct classify_point
{
  bbox box;
  int max_level;

  // Create the classifier
  classify_point(const bbox &b, int lvl) : box(b), max_level(lvl) {}

  // Classify a point
  inline __device__ __host__
  int operator()(const float2 &p) { return point_to_tag(p, box, max_level); }
};

void compute_tags(const thrust::device_vector<float2> &points, const bbox &bounds, int max_level, thrust::device_vector<int> &tags)
{
  thrust::transform(points.begin(), 
                    points.end(), 
                    tags.begin(), 
                    classify_point(bounds, max_level));
}


void sort_points_by_tag(thrust::device_vector<int> &tags, thrust::device_vector<int> &indices)
{
  thrust::sequence(indices.begin(), indices.end());
  thrust::sort_by_key(tags.begin(), tags.end(), indices.begin());
}


struct expand_active_nodes
{
  int level, max_level;
  const int *nodes;
  
  expand_active_nodes(int lvl, int max_lvl, const int *nodes) : level(lvl), max_level(max_lvl), nodes(nodes) {}
  
  inline __device__ __host__
  int operator()(int idx) const
  {
    int tag = nodes[idx/4];
    int mask = (idx&3) << (2*(max_level-level));
    return tag | mask;
  }
};

struct add
{
  typedef int result_type;
  int val;
  add(int v) : val(v) {}
  inline __device__ __host__ int operator()(int x) const { return x+val-1; }
};

struct mark_nodes
{
  int threshold;
  int last_level;
  
  mark_nodes(int threshold, int last_level) : threshold(threshold), last_level(last_level) {}

  template <typename tuple_type>
  inline __device__ __host__
  int operator()(const tuple_type &t) const
  {
    int lower_bound = thrust::get<0>(t);
    int upper_bound = thrust::get<1>(t);
    
    int count = upper_bound - lower_bound;
    if (count == 0)
    {
      return EMPTY;
    }
    else if (last_level || count < threshold)
    {
      return LEAF;
    }
    else
    {
      return NODE;
    }
  }
};

struct write_nodes
{
  int num_nodes, num_leaves;

  write_nodes(int num_nodes, int num_leaves) : 
    num_nodes(num_nodes), num_leaves(num_leaves) 
  {}

  template <typename tuple_type>
  inline __device__ __host__
  int operator()(const tuple_type &t) const
  {
    int node_type = thrust::get<0>(t);
    int node_idx  = thrust::get<1>(t);
    int leaf_idx  = thrust::get<2>(t);

    if (node_type == EMPTY)
    {
      return get_empty_id();
    }
    else if (node_type == LEAF)
    {
      return get_leaf_id(num_leaves + leaf_idx);
    }
    else
    {
      return num_nodes + 4 * node_idx;
    }
  }
};

struct make_leaf
{
  typedef int2 result_type;
  template <typename tuple_type>
  inline __device__ __host__
  int2 operator()(const tuple_type &t) const
  {
    int x = thrust::get<0>(t);
    int y = thrust::get<1>(t);

    return make_int2(x, y);
  }
};

void build_tree(const thrust::device_vector<int> &tags,
                const bbox &bounds,
                size_t max_level,
                int threshold,
                thrust::device_vector<int> &nodes,
                thrust::device_vector<int2> &leaves)
{
  thrust::device_vector<int> active_nodes(1,0);

  int num_nodes = 0, num_leaves = 0;

  // Build the tree one level at a time, starting at the root
  for (int level = 1 ; !active_nodes.empty() && level <= max_level ; ++level)
  {
    std::cout << "\n\n\n*************************\n";
    std::cout << "*** BUILDING LEVEL " << std::setw(4) << level << " *\n";
    std::cout << "*************************\n";

    // Number of nodes to process at this level
    int num_active_nodes = static_cast<int>(active_nodes.size());

    std::cout << "Active nodes:\n      ";
    for (int i = 1 ; i <= max_level ; ++i)
    {
      std::cout << "xy ";
    }
    std::cout << std::endl;
    for (int i = 0 ; i < num_active_nodes ; ++i)
    {
      std::cout << std::setw(4) << i << ": ";
      print_tag(active_nodes[i], max_level);
      std::cout << std::endl;
    }
    std::cout << std::endl;

    /******************************************
     * 6. Calculate children                  *
     ******************************************/

    // New children: 4 quadrants per active node = 4 children
    thrust::device_vector<int> children(4*num_active_nodes);

    // For each active node, generate the tag mask for each of its 4 children
    thrust::transform(thrust::make_counting_iterator(0),
                      thrust::make_counting_iterator(4*num_active_nodes),
                      children.begin(),
                      expand_active_nodes(level, max_level, thrust::raw_pointer_cast(&active_nodes.front())));

    std::cout << "Children:\n      ";
    for (int i = 1 ; i <= max_level ; ++i)
    {
      std::cout << "xy ";
    }
    std::cout << std::endl;
    for (int i = 0 ; i < children.size() ; ++i)
    {
      std::cout << std::setw(4) << i << ": ";
      print_tag(children[i], max_level);
      std::cout << std::endl;
    }
    std::cout << std::endl;

    /******************************************
     * 7. Determine interval for each child   *
     ******************************************/

    // For each child we need interval bounds
    thrust::device_vector<int> lower_bounds(children.size());
    thrust::device_vector<int> upper_bounds(children.size());

    // Locate lower and upper bounds for points in each quadrant
    thrust::lower_bound(tags.begin(),
                        tags.end(),
                        children.begin(),
                        children.end(),
                        lower_bounds.begin());

    add add_step(1 << 2*(max_level-level));
    thrust::upper_bound(tags.begin(),
                        tags.end(),
                        thrust::make_transform_iterator(children.begin(), add_step),
                        thrust::make_transform_iterator(children.end(), add_step),
                        upper_bounds.begin());

    std::cout << "Child bounds:\n      [ lower upper count ]\n";
    for (int i = 0 ; i < children.size() ; ++i)
    {
      std::cout << std::setw(4) << i << ": [ ";
      std::cout << std::setw(4) << lower_bounds[i] << "  ";
      std::cout << std::setw(4) << upper_bounds[i] << "  ";
      std::cout << std::setw(4) << upper_bounds[i] - lower_bounds[i] << "  ]";
      std::cout << std::endl;
    }
    std::cout << std::endl;

    /******************************************
     * 8. Mark each child as empty/leaf/node  *
     ******************************************/

    // Mark each child as either empty, a node, or a leaf
    thrust::device_vector<int> markers(children.size(), 0);

    thrust::transform(thrust::make_zip_iterator(
                          thrust::make_tuple(lower_bounds.begin(), upper_bounds.begin())),
                      thrust::make_zip_iterator(
                          thrust::make_tuple(lower_bounds.end(), upper_bounds.end())),
                      markers.begin(),
                      mark_nodes(threshold, level == max_level));

    std::cout << "Child markers:\n";
    for (int i = 0 ; i < children.size() ; ++i)
    {
      std::cout << std::setw(4) << i << ": [ ";
      std::cout << std::setw(5) << std::right;
      switch (markers[i])
      {
      case EMPTY:
        std::cout << "EMPTY ]";
        break;
      case LEAF:
        std::cout << "LEAF ]";
        break;
      case NODE:
        std::cout << "NODE ]";
        break;
      default:
        std::cout << "ERROR ]";
      }
      std::cout << std::endl;
    }
    std::cout << std::endl;

    /******************************************
     * 9. Enumerate nodes and leaves          *
     ******************************************/

    // Enumerate the nodes and leaves at this level
    thrust::device_vector<int> level_nodes(markers.size());
    thrust::device_vector<int> level_leaves(markers.size());

    // Enumerate nodes at this level
    thrust::transform_exclusive_scan(markers.begin(), 
                                     markers.end(), 
                                     level_nodes.begin(), 
                                     is_a<NODE>(), 
                                     0, 
                                     thrust::plus<int>());
    int num_level_nodes = level_nodes.back() + (markers.back() == NODE ? 1 : 0);

    // Enumerate leaves at this level
    thrust::transform_exclusive_scan(markers.begin(), 
                                     markers.end(), 
                                     level_leaves.begin(), 
                                     is_a<LEAF>(), 
                                     0, 
                                     thrust::plus<int>());
    int num_level_leaves = level_leaves.back() + (markers.back() == LEAF ? 1 : 0);

    std::cout << "Node/leaf enumeration:\n      [ nodeid leafid ]\n";
    for (int i = 0 ; i < children.size() ; ++i)
    {
      std::cout << std::setw(4) << i << ": [ ";
      switch (markers[i])
      {
      case EMPTY:
        std::cout << std::setw(4) << "." << "   " << std::setw(4) << "." << "   ]";
        break;
      case LEAF:
        std::cout << std::setw(4) << "." << "   " << std::setw(4) << level_leaves[i] << "   ]";
        break;
      case NODE:
        std::cout << std::setw(4) << level_nodes[i] << "   " << std::setw(4) << "." << "   ]";
        break;
      }
      std::cout << std::endl;
    }
    std::cout << std::endl;

    /******************************************
     * 10. Add the children to the node list  *
     ******************************************/

    // Add these children to the list of nodes
    nodes.resize(num_nodes + children.size());

    thrust::transform(thrust::make_zip_iterator(
                          thrust::make_tuple(
                              markers.begin(), level_nodes.begin(), level_leaves.begin())),
                      thrust::make_zip_iterator(
                          thrust::make_tuple(
                              markers.end(), level_nodes.end(), level_leaves.end())),
                      nodes.begin() + num_nodes,
                      write_nodes(num_nodes + 4 * num_active_nodes, num_leaves));

    // Update the number of nodes
    num_nodes += 4 * num_active_nodes;

    print_nodes(nodes);

    /******************************************
     * 11. Add the leaves to the leaf list    *
     ******************************************/

    // Add child leaves to the list of leaves
    leaves.resize(num_leaves + num_level_leaves);
    thrust::scatter_if(thrust::make_transform_iterator(
                           thrust::make_zip_iterator(
                               thrust::make_tuple(lower_bounds.begin(), upper_bounds.begin())),
                           make_leaf()),
                       thrust::make_transform_iterator(
                           thrust::make_zip_iterator(
                               thrust::make_tuple(lower_bounds.end(), upper_bounds.end())),
                           make_leaf()),
                       level_leaves.begin(),
                       markers.begin(),
                       leaves.begin() + num_leaves,
                       is_a<LEAF>());

    // Update the number of leaves
    num_leaves += num_level_leaves;

    print_leaves(leaves);

    /******************************************
     * 12. Set the nodes for the next level   *
     ******************************************/
    
    // Set active nodes for the next level to be all the childs nodes from this level
    active_nodes.resize(num_level_nodes);

    thrust::copy_if(children.begin(),
                    children.end(),
                    markers.begin(),
                    active_nodes.begin(),
                    is_a<NODE>());

    // Update the number of active nodes.
    num_active_nodes = num_level_nodes;
  }
}

int main()
{
  const size_t num_points = 12;
  const int threshold = 2; // A node with fewer than threshold points is a leaf.
  const int max_level = 3;

  thrust::device_vector<float2> points(num_points);

  /******************************************
   * 1. Generate points                     *
   ******************************************/

  // Generate random points using Thrust
  thrust::tabulate(points.begin(), points.end(), random_point());

  std::cout << "Points:\n";
  for (int i = 0 ; i < points.size() ; ++i)
  {
    std::cout << std::setw(4) << i << " " << points[i] << std::endl;
  }
  std::cout << std::endl;

  /******************************************
   * 2. Compute bounding box                *
   ******************************************/

  bbox bounds = compute_bounding_box(points);

  float xmid = 0.5f * (bounds.xmin + bounds.xmax);
  float ymid = 0.5f * (bounds.ymin + bounds.ymax);
  std::cout << "Bounding box:\n";
  std::cout << "   min: " << make_float2(bounds.xmin, bounds.ymin) << std::endl;
  std::cout << "   mid: " << make_float2(xmid, ymid) << std::endl;
  std::cout << "   max: " << make_float2(bounds.xmax, bounds.ymax) << std::endl;
  std::cout << std::endl;

  /******************************************
   * 3. Classify points                     *
   ******************************************/

  thrust::device_vector<int> tags(num_points);
  
  compute_tags(points, bounds, max_level, tags);

  std::cout << "Tags:                       ";
  for (int level = 1 ; level <= max_level ; ++level)
  {
    std::cout << std::setw(3) << std::left << level;
  }
  std::cout << "\n                            ";
  for (int level = 1 ; level <= max_level ; ++level)
  {
    std::cout << "xy ";
  }
  std::cout << std::right << std::endl;
  for (int i = 0 ; i < points.size() ; ++i)
  {
    int tag = tags[i];
    std::cout << std::setw(4) << i << " " << points[i] << ":  ";
    print_tag(tags[i], max_level);
    std::cout << std::endl;
  }
  std::cout << std::endl;
  
  /******************************************
   * 4. Sort according to classification    *
   ******************************************/

  thrust::device_vector<int> indices(num_points);

  // Now that we have the geometric information, we can sort the
  // points accordingly.
  sort_points_by_tag(tags, indices);

  std::cout << "Sorted tags:                ";
  for (int level = 1 ; level <= max_level ; ++level)
  {
    std::cout << std::setw(3) << std::left << level;
  }
  std::cout << "\n                            ";
  for (int level = 1 ; level <= max_level ; ++level)
  {
    std::cout << "xy ";
  }
  std::cout << std::right << std::endl;
  for (int i = 0 ; i < points.size() ; ++i)
  {
    int tag = tags[i];
    std::cout << std::setw(4) << i << " " << points[i] << ":  ";
    print_tag(tags[i], max_level);
    std::cout << "  original index " << std::setw(4) << indices[i] << std::endl;
  }
  std::cout << std::endl;

  /******************************************
   * 5. Build the tree                      *
   ******************************************/

  thrust::device_vector<int> nodes;
  thrust::device_vector<int2> leaves;
  
  build_tree(tags, bounds, max_level, threshold, nodes, leaves);

  return 0;
}


